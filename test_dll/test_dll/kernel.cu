#include "hip/hip_runtime.h"
#include"cuda_kernel.cuh"
#include<cuda_gl_interop.h>
#include<iostream>
#include<fstream>


__global__ void kernel(hipSurfaceObject_t surf1, hipSurfaceObject_t surf2, int szer, int wys, bool flaga) {

	register int wsp_x = blockIdx.x * blockDim.x + threadIdx.x;
	register int wsp_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (wsp_x < szer && wsp_y < wys) {

	//	register int i = wsp_y*szer + wsp_x;
	//	uchar4 voxel;// = make_uchar4(255, 255, 255, 255);

	//	voxel.x = (flaga) ? 255 : 1;
	//	voxel.y = (flaga) ? 255 : 1;
	//	voxel.z = (flaga) ? 255 : 1;
	//	voxel.w = (flaga) ? 255 : 1;
	//	surf2Dwrite(voxel, (flaga) ? surf2 : surf1, wsp_x * sizeof(uchar4), wsp_y);


		/*
		register int i = wsp_y*szer + wsp_x;
		uchar4 voxel;// = make_uchar4(255, 255, 255, 255);
		//	surf2Dread(&voxel, surf[blockIdx.x], wsp_x * sizeof(uchar4), wsp_y);
		surf2Dread(&voxel, (flaga)?surf1:surf2, wsp_x * sizeof(uchar4), wsp_y);
		//if (voxel.x == 255) {

		voxel.x = (flaga) ? 255 : 1;
		voxel.y = (flaga) ? 255 : 1;
		voxel.z = (flaga) ? 255 : 1;
		voxel.w = (flaga) ? 255 : 1;
		//surf2Dwrite(voxel, surf[blockIdx.x], wsp_x * sizeof(uchar4), wsp_y);
		surf2Dwrite(voxel, (flaga) ? surf2 : surf1, wsp_x * sizeof(uchar4), wsp_y);
		//}
		*/
	}
}

typedef unsigned int  uint;
typedef unsigned char uchar;

void cuda_texture_interface::zarejestrujTeksture(bool flaga, int indeks, GLuint image, GLenum target, unsigned int  flags, unsigned int arrayIndex, unsigned int mipLevel, unsigned long* tekstura, int szer, int wys) {

	wysokosc = wys;
	szerokosc = szer;

	if (flaga) {

		HANDLE_ERROR(hipGraphicsGLRegisterImage(&r[indeks], image, target, flags));
		HANDLE_ERROR(hipGraphicsMapResources(1, &r[indeks], stream));
		HANDLE_ERROR(hipGraphicsSubResourceGetMappedArray(&tab[indeks], r[indeks], arrayIndex, mipLevel));

		lalala(indeks, tekstura, szer, wys, true);


	}
	else {

		HANDLE_ERROR(hipGraphicsGLRegisterImage(&r2[indeks], image, target, flags));
		HANDLE_ERROR(hipGraphicsMapResources(1, &r2[indeks], stream));
		HANDLE_ERROR(hipGraphicsSubResourceGetMappedArray(&tab2[indeks], r2[indeks], arrayIndex, mipLevel));
		lalala(indeks, tekstura, szer, wys, false);

	}
}


void cuda_texture_interface::odmapowanieTekstur() {

	HANDLE_ERROR(hipGraphicsUnmapResources(rozmiar, r, stream));
	HANDLE_ERROR(hipGraphicsUnmapResources(rozmiar, r2, stream));
}

void cuda_texture_interface::inicjalizacja() {

	hipDeviceProp_t prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	HANDLE_ERROR(cudaGLSetGLDevice(dev));
	hipStreamCreateWithFlags(&stream, hipStreamDefault);
}

void cuda_texture_interface::lalala(int i, unsigned long* tekstura, int szer, int wys, bool flaga) {

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	
	HANDLE_ERROR(hipMemcpyToArray((flaga) ? tab[i] : tab2[i], 0, 0, tekstura, szer*wys * sizeof(unsigned long), hipMemcpyHostToDevice));
	
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));

	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = (flaga) ? tab[i] : tab2[i];
	
	HANDLE_ERROR(hipCreateSurfaceObject(flaga ? &tex[i] : &tex2[i], &resDesc));
}

void cuda_texture_interface::akcja(bool flaga) {

	dim3 block(32, 32);
	dim3 grid((szerokosc - 1) / 32 + 1, (szerokosc - 1) / 32 + 1);

	for (int i = 0; i<rozmiar; ++i)
		kernel << <grid, block >> > (tex[i], tex2[i], szerokosc, wysokosc, flaga);

}