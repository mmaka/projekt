#include "hip/hip_runtime.h"
#include"cuda_opengl_interop.cuh"
#include<cuda_gl_interop.h>


__global__ void kernel(hipSurfaceObject_t surf1, hipSurfaceObject_t surf2, int szer, int wys, bool flaga) {

	register int wsp_x = blockIdx.x * blockDim.x + threadIdx.x;
	register int wsp_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (wsp_x < szer && wsp_y < wys) {

		//	register int i = wsp_y*szer + wsp_x;
		//	uchar4 voxel;
		//surf2Dread(&voxel, (flaga) ? surf1 : surf2, wsp_x * sizeof(uchar4), wsp_y);
		//	surf2Dwrite(voxel, (flaga) ? surf2 : surf1, wsp_x * sizeof(uchar4), wsp_y);
	}
}



void Cuda_OpenGL_Interop::zarejestrujTekstury(bool flaga, int indeks, GLuint image, GLenum target, unsigned int  flags, unsigned int arrayIndex, unsigned int mipLevel, unsigned long* tekstura) {

	if (flaga) {

		HANDLE_ERROR(hipGraphicsGLRegisterImage(&r1[indeks], image, target, flags));
		HANDLE_ERROR(hipGraphicsMapResources(1, &r1[indeks], strumien));
		HANDLE_ERROR(hipGraphicsSubResourceGetMappedArray(&tab1[indeks], r1[indeks], arrayIndex, mipLevel));
	//	pobieranie_danych(indeks, tekstura, true);//pobieranie danych b�dzie zorganizowane zupe�nie inaczej - b�dzie niezale�ne od rejestracji - p�ki co na potrzeby przyk�adu dzia�ania

	}
	else {

		HANDLE_ERROR(hipGraphicsGLRegisterImage(&r2[indeks], image, target, flags));
		HANDLE_ERROR(hipGraphicsMapResources(1, &r2[indeks], strumien));
		HANDLE_ERROR(hipGraphicsSubResourceGetMappedArray(&tab2[indeks], r2[indeks], arrayIndex, mipLevel));
	//	pobieranie_danych(indeks, tekstura, false);

	}
}

void Cuda_OpenGL_Interop::rejestrowanieTekstur(GLuint *indeksy, hipGraphicsResource_t *res, hipArray_t *arr, int ileTekstur,hipStream_t strumien, GLenum target, unsigned int  flags, unsigned int arrayIndex, unsigned int mipLevel) {
/*
	for (int i = 0, end = ileTekstur; i != end; ++i)
		HANDLE_ERROR(hipGraphicsGLRegisterImage(&res[i], indeksy[i], target, flags));

	HANDLE_ERROR(hipGraphicsMapResources(ileTekstur, res, strumien));

	for(int i = 0, end = ileTekstur;i!=end;++i)
		HANDLE_ERROR(hipGraphicsSubResourceGetMappedArray(&arr[i], res[i], arrayIndex, mipLevel));
		*/
	for (int i = 0, end = ileTekstur; i != end; ++i) {

		HANDLE_ERROR(hipGraphicsGLRegisterImage(&res[i], indeksy[i], target, flags));
		HANDLE_ERROR(hipGraphicsMapResources(1, &res[i], strumien));
		HANDLE_ERROR(hipGraphicsSubResourceGetMappedArray(&arr[i], res[i], arrayIndex, mipLevel));
	}

}

void Cuda_OpenGL_Interop::odmapowanieTekstur(hipGraphicsResource_t *res, int ileTekstur, hipStream_t stream) {

	HANDLE_ERROR(hipGraphicsUnmapResources(ileTekstur, res, stream));

}

void Cuda_OpenGL_Interop::odmapowanieTekstur2() {

	HANDLE_ERROR(hipGraphicsUnmapResources(rozmiar, r1, strumien));
	HANDLE_ERROR(hipGraphicsUnmapResources(rozmiar, r2, strumien));
}


void Cuda_OpenGL_Interop::init() {

	hipDeviceProp_t prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	HANDLE_ERROR(cudaGLSetGLDevice(dev));
	hipStreamCreateWithFlags(&strumien, hipStreamDefault);
}
/*
void Cuda_OpenGL_Interop::pobieranie_danych(int i, unsigned long* tekstura, bool flaga) {

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);// to powinno by� ustalone podczas tworzenia instancji klasy, bo te ustawienia si� nie zmieniaj�

	HANDLE_ERROR(hipMemcpyToArray((flaga) ? tab1[i] : tab2[i], 0, 0, tekstura, szer_tekstury*wys_tekstury * sizeof(unsigned long), hipMemcpyHostToDevice));

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));

	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = (flaga) ? tab1[i] : tab2[i];

	HANDLE_ERROR(hipCreateSurfaceObject(flaga ? &surf1[i] : &surf2[i], &resDesc));
}
*/

void Cuda_OpenGL_Interop::launch(bool flaga) {

	dim3 block(32, 32);
	dim3 grid((szer_tekstury - 1) / 32 + 1, (szer_tekstury - 1) / 32 + 1);

	for (int i = 0; i<rozmiar; ++i)
		kernel << <grid, block >> > (surf1[i], surf2[i], szer_tekstury, wys_tekstury, flaga);//to b�dzie realizowane przy u�yciu strumieni - surface'y zapisane s� w tablicy, ale nie mo�na dzia�a� na nich r�wnolegle - przynajmniej p�ki co mi si� nie uda�o

}


///////////////////////////////////////////////


void PojemnikNaTekstury::pobieranie_danych(int i, unsigned long* tekstura, bool flaga) {

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);// to powinno by� ustalone podczas tworzenia instancji klasy, bo te ustawienia si� nie zmieniaj�

	HANDLE_ERROR(hipMemcpyToArray((flaga) ? tab1[i] : tab2[i], 0, 0, tekstura, szer_tekstury*wys_tekstury * sizeof(unsigned long), hipMemcpyHostToDevice));

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));

	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = (flaga) ? tab1[i] : tab2[i];

	HANDLE_ERROR(hipCreateSurfaceObject(flaga ? &surf1[i] : &surf2[i], &resDesc));
}

void PojemnikNaTekstury::wpiszTekstureWCudaArray() {

	//int szer, wys;
	unsigned long *tekstura = WczytajObrazZPlikuBitmap(NULL, "kwadrat_czer.bmp", szer_tekstury, wys_tekstury, false, 255);
	for (int i = 0; i < liczbaTekstur; ++i) {

		pobieranie_danych(i, tekstura, true);
		pobieranie_danych(i, tekstura, false);
	}
		


}

void PojemnikNaTekstury::surf_init() {

	for (int i = 0, end = liczbaTekstur; i != end; ++i) {

		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);// to powinno by� ustalone podczas tworzenia instancji klasy, bo te ustawienia si� nie zmieniaj�
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = tab1[i];
		hipSurfaceObject_t surf;
		HANDLE_ERROR(hipCreateSurfaceObject(&surf, &resDesc));
		kolejka1.push(surf);

	}

	for (int i = 0, end = liczbaTekstur; i != end; ++i) {

		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);// to powinno by� ustalone podczas tworzenia instancji klasy, bo te ustawienia si� nie zmieniaj�
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = tab2[i];
		hipSurfaceObject_t surf;
		HANDLE_ERROR(hipCreateSurfaceObject(&surf, &resDesc));
		kolejka2.push(surf);

	}
}

__global__ void wpisz_teksture(hipSurfaceObject_t surf,uchar4 *tex, int szer, int wys) {

	register int wsp_x = blockIdx.x * blockDim.x + threadIdx.x;
	register int wsp_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (wsp_x < szer && wsp_y < wys) {

		surf2Dwrite(tex[wsp_y*szer + wsp_x],surf, wsp_x * sizeof(uchar4), wsp_y);
		//	register int i = wsp_y*szer + wsp_x;
		//	uchar4 voxel;
		//surf2Dread(&voxel, (flaga) ? surf1 : surf2, wsp_x * sizeof(uchar4), wsp_y);
		//	surf2Dwrite(voxel, (flaga) ? surf2 : surf1, wsp_x * sizeof(uchar4), wsp_y);
	}
}

void PojemnikNaTekstury::launch(const hipSurfaceObject_t& surf,const pakiet_danych& p) {

	dim3 block(32, 32);
	dim3 grid((p.rozmiar_x - 1) / 32 + 1, (p.rozmiar_y - 1) / 32 + 1);

	wpisz_teksture << <grid, block >> > (surf,p.ptr, p.rozmiar_x, p.rozmiar_y);//to b�dzie realizowane przy u�yciu strumieni - surface'y zapisane s� w tablicy, ale nie mo�na dzia�a� na nich r�wnolegle - przynajmniej p�ki co mi si� nie uda�o

}


void PojemnikNaTekstury::old_dostawcaTekstur(const pakiet_danych& p,bool flaga) {

	hipSurfaceObject_t tmp = (flaga)?kolejka2.front():kolejka1.front();
	(flaga)?kolejka2.pop():kolejka1.pop();
	launch(tmp, p);
	(flaga)?kolejka2.push(tmp):kolejka1.push(tmp);
	pierwszy = (--pierwszy) % 300;
	//HANDLE_ERROR(hipFree(p.ptr));

}


void PojemnikNaTekstury::dostawcaTekstur(const pakiet_danych& p) {

	mutex().lock();
	tasma_produkcyjna.push(p);
	mutex().unlock();
	//hipSurfaceObject_t tmp = (flaga)?kolejka2.front():kolejka1.front();
	//(flaga)?kolejka2.pop():kolejka1.pop();
	//launch(tmp, p);
	//(flaga)?kolejka2.push(tmp):kolejka1.push(tmp);
	//HANDLE_ERROR(hipFree(p.ptr));
	
}

void PojemnikNaTekstury::producentTekstur() {


	std::unique_lock<std::mutex> lock(mutex());
	cond_v().wait(lock);
	bool flaga = true;
	hipSurfaceObject_t tmp = (flaga) ? kolejka2.front() : kolejka1.front();
	pakiet_danych pak = tasma_produkcyjna.front();
	tasma_produkcyjna.pop();
	launch(tmp, pak);
	(flaga) ? kolejka2.push(tmp) : kolejka1.push(tmp);
}

void PojemnikNaTekstury::przekazWskaznik(pakiet_danych& p) {

	int szer = 0, wys = 0;
	unsigned long* tekstura = WczytajObrazZPlikuBitmap(NULL, "kwadrat_czer.bmp", szer, wys, false, 255);
	uchar4* ptr;
	HANDLE_ERROR(hipMalloc(&ptr, szer*wys * sizeof(unsigned long)));
	HANDLE_ERROR(hipMemcpy(ptr, tekstura, szer*wys * sizeof(unsigned long), hipMemcpyHostToDevice));
//	p.ptr = ptr;
	p.rozmiar_x = szer;
	p.rozmiar_y = wys;

}

void PojemnikNaTekstury::generujIndeksy(std::vector<unsigned int>& v) {

	v.reserve(300);
	
	for (unsigned int i = pierwszy; i < 300; ++i) v.push_back(i);

	for (unsigned int i = 0, end = 300 - v.size(); i != end; ++i) v.push_back(i);


}